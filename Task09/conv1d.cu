#include "hip/hip_runtime.h"
#include <iostream>
#include "utils.h"

// Assumptions : 
// KERNEL_WIDTH is small and can fit in constant memory
// KERNEL_WIDTH is odd

const int INPUT_TILE_WIDTH = 256;
const int KERNEL_WIDTH = 5;
__constant__ float kernel[KERNEL_WIDTH];

__global__ void conv1d_unoptim(const float *a, const float *b, float *c, int N, int K)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N - K + 1)
    {
        float sum = 0.0f;
        for (int i = 0; i < K; i++)
        {
            sum += a[idx + i] * b[i];
        }
        c[idx] = sum;
    }
}

__global__ void conv1d(const float *in, float *out, int N)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int outputTileSize = INPUT_TILE_WIDTH - 2 * (KERNEL_WIDTH/2);
    int radius = KERNEL_WIDTH/2;
    int inputIdx = bx * outputTileSize + tx - radius;

    __shared__ float sharedMem[INPUT_TILE_WIDTH];

    if (0 <= inputIdx && inputIdx < N) 
    {
        sharedMem[tx] = in[inputIdx];
    }
    else 
    {
        sharedMem[tx] = 0.0f;
    }
    __syncthreads();

    if (radius <= inputIdx && inputIdx < N - radius)
    {
        if (0 <= tx-radius && tx-radius < outputTileSize)
        {
            float outValue = 0.0f;
            for (int k = 0; k < KERNEL_WIDTH; k++)
            {
                outValue += kernel[k] * sharedMem[tx - radius + k];
            }
            out[inputIdx-radius] = outValue;
        }
    }
}

__global__ void conv1d_output_tile(const float *in, float *out, int N)
{
    int tx = threadIdx.x, bx = blockIdx.x, radius = (KERNEL_WIDTH/2);
    int inputTileSize = OUTPUT_TILE_WIDTH + 2 * radius;
    int outputIdx = bx * OUTPUT_TILE_WIDTH + tx;

    __shared__ float sharedMem[inputTileSize];

    if (outputIdx < radius || outputIdx >= N-radius)
    {
        return;
    }

    if (tx < radius)
    {
        sharedMem[tx] = in[outputIdx-radius];
    }
    if (OUTPUT_TILE_WIDTH-tx >= radius)
    {
        sharedMem[tx+2*radius] = in[outputIdx+radius];
    }
    sharedMem[tx+radius] = in[outputIdx];
    __syncthreads();

    float pValue = 0.0f;
    for (int k=0; k<KERNEL_WIDTH; k++)
    {
        pValue += kernel[k] * sharedMem[tx+k];
    }
    out[outputIdx-radius] = pValue;
}

void launch_conv1d(const float *a, const float *b, float *c, int N){
    float *ad, *cd;
    hipMalloc((void**)&ad, N * sizeof(float));
    hipMalloc((void **)&cd, (N - KERNEL_WIDTH + 1) * sizeof(float));

    hipMemcpy(ad, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel), b, KERNEL_WIDTH * sizeof(float));

    int blockSize = INPUT_TILE_WIDTH;
    int gridSize = (N + blockSize - 1) / blockSize;

    conv1d<<<gridSize, blockSize>>>(ad, cd, N);
    hipDeviceSynchronize();

    hipMemcpy(c, cd, (N - KERNEL_WIDTH + 1)*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(ad);
    hipFree(cd);
}

void launch_conv1d_unoptim(const float *a, const float *b, float *c, int N, int K)
{
    float *ad, *bd, *cd;
    hipMalloc((void **)&ad, N * sizeof(float));
    hipMalloc((void **)&bd, K * sizeof(float));
    hipMalloc((void **)&cd, (N - K + 1) * sizeof(float));

    hipMemcpy(ad, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bd, b, K * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N - K + 1 + blockSize - 1) / blockSize;

    conv1d_unoptim<<<gridSize, blockSize>>>(ad, bd, cd, N, K);
    hipDeviceSynchronize();

    hipMemcpy(c, cd, (N - K + 1) * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
}

int main()
{
    int N = 10;
    int K = KERNEL_WIDTH;
    float a[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float b[K] = {2, 1, 0, -1, -2};
    float c[N - K + 1];
    float c_optim[N - K + 1] = {0.0f};

    launch_conv1d_unoptim(a, b, c, N, K);

    launch_conv1d(a, b, c_optim, N);

    std::cout << "Convolution result : ";
    print_array(c, N - KERNEL_WIDTH + 1);

    std::cout << "Convolution optimised result: ";
    print_array(c_optim, N - KERNEL_WIDTH + 1);

    return 0;
}