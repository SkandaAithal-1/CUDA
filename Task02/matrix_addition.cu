#include "hip/hip_runtime.h"
#include <iostream>
#include "kernels.cuh"

__global__ void matrix_addition(const float *A, const float *B, float *C, int M, int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N)
    {
        C[row * N + col] = A[row * N + col] + B[row * N + col];
    }
}

int main()
{
    const int M = 1024;
    const int N = 1024;
    float A[M * N], B[M * N], C[M * N];
    float *Ad, *Bd, *Cd;
    size_t size = M * N * sizeof(float);

    // Allocate memory on the device
    hipMalloc((void **)&Ad, size);
    hipMalloc((void **)&Bd, size);
    hipMalloc((void **)&Cd, size);

    // Copy data from host to device
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    hipMemcpy(Cd, C, size, hipMemcpyHostToDevice);

    // Execution parameter configuration
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil((float)M / blockDim.x), ceil((float)N / blockDim.y));

    // Launch kernel
    matrix_addition<<<gridDim, blockDim>>>(Ad, Bd, Cd, M, N);
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);

    return 0;
}