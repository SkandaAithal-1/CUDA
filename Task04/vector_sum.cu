#include "hip/hip_runtime.h"
#include<iostream>

__global__ void vector_sum(const float *x, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = N >> 1;
    if (idx < stride)
    {
        x[idx] += x[idx + stride];
    }
}

int main()
{
    int N = 8;
    float x[N] = [1, 2, 3, 4, 5, 6, 7, 8];
    float y;

    hipMalloc((void **)&xd, N * sizeof(float));

    hipMemcpy(xd, x, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    for (int size = N; size > 0; size >>= 1){
        vector_sum<<<gridSize, blockSize>>>(xd, size);
    }

    hipDeviceSynchronize();

    hipMemcpy(y, xd, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Sum : " << y << std::endl;
}