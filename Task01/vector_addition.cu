#include "hip/hip_runtime.h"
#include<iostream>

__global__ void vector_addition(const float *a, const float *b, float *c, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        c[idx] = a[idx] + b[idx];
    }
}

int main(){
    int N = 100;
    float A[n], B[n], C[n];
    float *Ad, *Bd, *Cd;

    //Allocate memory on the device 
    hipMalloc((void**)&Ad, N * (sizeof(float)));
    hipMalloc((void**)&Bd, N * (sizeof(float)));
    hipMalloc((void**)&Cd, N * (sizeof(float)));

    // Copy data from host to device
    hipMemcpy(Ad, A, N * (sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, N * (sizeof(float)), hipMemcpyHostToDevice);

    // Configure execution parameters
    int blockSize = 256;
    int gridSize = ceil(N / blockSize);  

    // Launch kernel
    vector_addition<<<gridSize, blockSize>>>(Ad, Bd, Cd, N);

    // Copy result from device to host
    hipMemcpy(C, Cd, N * (sizeof(float)), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);

    return 0;
}