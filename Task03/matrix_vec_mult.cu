#include "hip/hip_runtime.h"
#include <iostream>

__global__ void matrix_vector_mult(const float *A, const float *x, float* y, int M, int N){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M){
        for (int i=0; i<N; i++){
            y[row] += A[row * N + i] * x[i];
        }
    }
}

__global__ void matrix_vector_mult_optim(const float *A, const float *x, float *y, int M, int N){
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    // int TILE_WIDTH = bx;
    int row = bx * blockDim.x + tx;
    if (row >= M) return;

    __shared__ float x_shared[bx]; // bx = TILE_WIDTH 
    float sum = 0;
    for (int phase=0; phase < (int)(N/bx); phase++){  // bx = TILE_WIDTH
        // Load x into shared memory
        if (phase * bx + tx < N){
            x_shared[tx] = x[phase * bx + tx]; // bx = TILE_WIDTH
        }else{
            x_shared[tx] = 0;
        }
        __syncthreads();

        // Accumulate the sum for this phase
        for (int i=0; i<bx; i++){ // bx = TILE_WIDTH
            if (phase * bx + i >= N) break;
            sum += A[row * N + phase * bx + i] * x_shared[i]; // bx = TILE_WIDTH
        }
        __syncthreads();
    }
    y[row] = sum;
}

int main()
{
    int M = 100, N = 100;
    float A[M*N], x[N], y[M], y_optim[M];
    float *Ad, *xd, *yd, *y_optimd;
    for (int i=0; i<M*N; i++){A[i] = 1.0f;}
    for (int i=0; i<N; i++){x[i] = 1.0f;}

    // Allocate device memory
    hipMalloc((void**)&Ad, M*N*sizeof(float));
    hipMalloc((void**)&xd, N*(sizeof(float)));
    hipMalloc((void**)&yd, M*(sizeof(float)));
    hipMalloc((void**)&y_optimd, M*(sizeof(float)));

    // Copy data from host to device
    hipMemcpy(Ad, A, M*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(xd, x, N*sizeof(float), hipMemcpyHostToDevice);

    // Execution parameters configuration
    int blockSize = 256;
    int gridSize = ceil(M / blockSize);

    // Launch kernel
    matrix_vector_mult<<<gridSize, blockSize>>>(Ad, xd, yd, M, N);
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(y, yd, M*sizeof(float), hipMemcpyDeviceToHost);

    // Launch optimized kernel
    matrix_vector_mult_optim<<<gridSize, blockSize>>>(Ad, xd, y_optimd, M, N);
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(y_optim, y_optimd, M*sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(Ad);
    hipFree(xd);
    hipFree(yd);
    hipFree(y_optimd);

    return 0;
}